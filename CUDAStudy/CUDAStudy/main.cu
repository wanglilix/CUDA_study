#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include "" 
#include <stdio.h>
#include<time.h>

#define DATA_SIZE 19484
#define BLOCK_NUM 32
#define THREAD_NUM 256
//��Ҫ��һ��cpp�ļ�������__device__��__global__����
__global__ void sumOfSquares(int *num, int* result)
{
	extern __shared__ int shared[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
	int sum = 0;
	int i;
	shared[tid] = 0;
	for (i = bid * THREAD_NUM + tid; i < DATA_SIZE;i += BLOCK_NUM * THREAD_NUM) 
	{
		shared[tid] += num[i] * num[i];
	}
	__syncthreads();
	if (tid == 0) 
	{
		result[bid] = shared[0];
	}
}
extern "C"  hipError_t USECUDA(int* data, int& sum)
{
	int* gpudata;
	int* result;
	hipError_t cudaStatus;

	cudaStatus = hipMalloc((void**)&gpudata, sizeof(int)* DATA_SIZE);
	cudaStatus = hipMalloc((void**)&result, sizeof(int)*BLOCK_NUM);

	cudaStatus = hipMemcpy(gpudata, data, DATA_SIZE * sizeof(int), hipMemcpyHostToDevice);


	sumOfSquares << <BLOCK_NUM, THREAD_NUM, THREAD_NUM*sizeof(int) >> >(gpudata, result);//ע�⹲���ڴ���Ϊ0ʱ ��������ǲ�����

	int sum2[BLOCK_NUM];
	hipMemcpy(sum2, result, sizeof(int)*BLOCK_NUM, hipMemcpyDeviceToHost);
	for (int i = 0; i < BLOCK_NUM; i++) {
		sum += sum2[i];
	}

	cudaStatus = hipGetLastError();

	hipFree(gpudata);
	hipFree(result);
	return cudaStatus;
}